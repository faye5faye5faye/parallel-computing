#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <locale.h>

unsigned int getmax(unsigned int *, unsigned int);
unsigned int getmaxcu(unsigned int *, unsigned int);




__global__ void helper(unsigned int * d, unsigned int * comp, int size) {
  int tid; 
  tid = threadIdx.x;

  int i; 
  i = blockIdx.x * blockDim.x + threadIdx.x;

  if(i < size){
    for(uint stride = 1; stride < blockDim.x; stride *= 2){
      if(i % (2 * stride) == 0){
        if(comp[i] < comp[i + stride]){
          comp[i] = comp[i + stride]; 
        }
      } 
      __syncthreads();  
    }


    if(tid == 0){
      if(comp[blockIdx.x * blockDim.x] > d[blockIdx.x]){
        d[blockIdx.x] = comp[blockIdx.x * blockDim.x]; 
      }
    }
  }
} 


unsigned int getmaxcu(unsigned int num[], unsigned int size){
  uint *x, *y;

  hipMalloc((void**)&x, size*sizeof(uint));
  hipMalloc((void**)&y, size*sizeof(uint));

  hipMemcpy(x, num, sizeof(uint) * size, hipMemcpyHostToDevice); 
  hipMemcpy(y, num, sizeof(uint) * size, hipMemcpyHostToDevice); 

  int a = (int)((size - size % 1024)/1024) + 1;

  helper<<<a, 1024>>>(x, y, size);

  hipMemcpy(num, x, sizeof(uint) * size, hipMemcpyDeviceToHost); 

  uint max = 0; 
  for(int i = 0; i < a + 1; i++){
    max = max > num[i] ? max : num[i]; 
  }

  hipFree(x); 
  hipFree(y); 

  return(max);
}


int main(int argc, char *argv[])
{
    unsigned int size = 0;  // The size of the array
    unsigned int i;  // loop index
    unsigned int * numbers; //pointer to the array
    
    if(argc !=2)
    {
       printf("usage: maxseq num\n");
       printf("num = size of the array\n");
       exit(1);
    }
   
    size = atol(argv[1]);

    numbers = (unsigned int *)malloc(size * sizeof(unsigned int));
    if( !numbers ){
       printf("Unable to allocate mem for an array of size %u\n", size);
       exit(1);
    }    

    srand(time(NULL)); // setting a seed for the random number generator
    // Fill-up the array with random numbers from 0 to size-1 
    for( i = 0; i < size; i++)
       numbers[i] = rand()  % size;    
   
    printf("The maximum number in the array is: %u\n", getmaxcu(numbers, size));

    free(numbers);
    exit(0);
}







